#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void compare(int *in_d, int* out_d)
{
	if (in_d[blockIdx.x] == 6)
	{
		out_d[blockIdx.x] = 1;
	}
	else
		out_d[blockIdx.x] = 0;
}

#define SIZE 16


int main()
{
	//create two arrays
	size_t bytes = SIZE * sizeof(int);
	int *input, *output;
	input = (int*)malloc(bytes);
	output = (int*)malloc(bytes);

	//fill the input array with random values in [0,9]
	srand(time(0));
	int i;
	for (i = 0; i < SIZE; i++)
	{
		input[i] = rand() % 10;
		output[i] = -1;
	}

	//now allocate both arrays on the gpu
	int *input_d, *output_d;
	hipMalloc((void**)&input_d, bytes);
	hipMalloc((void**)&output_d, bytes);

	//now we copy the input array to the gpu
	hipMemcpy(input_d, input, bytes, hipMemcpyHostToDevice);

	//launch the kernel
	compare << <SIZE, 1 >> >(input_d, output_d);

	//copy the output array back to the cpu mem
	hipMemcpy(output, output_d, bytes, hipMemcpyDeviceToHost);

	//free memory on the gpu
	hipFree(input_d);
	hipFree(output_d);
	input_d = 0;
	output_d = 0;

	//display our answers
	int total = 0;
	for (int i = 0; i < SIZE; i++)
	{
		if (output[i] == 1)
			total += 1;
	}

	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", input[i]);
	}
	printf("\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", output[i]);
	}

	printf("total sixes = %d\n", total);

	//free our arrays from memory
	free(input);
	free(output);
	input = 0;
	output = 0;

}


